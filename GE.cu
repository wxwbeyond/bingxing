#include "hip/hip_runtime.h"
#pragma warning(disable : 4996)
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include"mybitset.h"
using namespace std;

const int BLOCK_SIZE = 1024;
int** eks_bits = new int* [column_num_c];
int** ets_bits = new int* [et_num_c];
//CUDA�㷨�˺���
__global__ void upshift_kernel(int i, int** gpu_eks, int** gpu_ets, int* gpu_lp_ets, int et_num_c) 
{
	//���߳���
	if (blockIdx.x == 0 && threadIdx.x == 0)
		for (int j = 0; j < et_num_c; j++)
		{
			if (i == gpu_lp_ets[j])  // ˵�����ڶ�Ӧ����Ԫ��
			{
				gpu_eks[i] = gpu_ets[j];
				gpu_lp_ets[j] = -1;
				break;
			}
		}
}
__global__ void elim_kernel(int i, int** gpu_ek, int** gpu_et, int* gpu_lp_ets, int et_num_c, int column_num_c, int bit_size)
{
	int tx = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockIdx.x;//ÿ���鸺��һ������Ԫ��
	bool find = false;
	for (int j = row; j < et_num_c; j += blockDim.x)  // ѭ�����ֲ��л�
	{
		if (i == gpu_lp_ets[j])  // ˵�����ڶ�Ӧ����Ԫ��
		{
			for (int k = 0; k < bit_size; k++)
				gpu_et[j][k] ^= gpu_ek[i][k];
			__syncthreads();
			if (threadIdx.x == 0)
			{
				for (int k = 0; k < bit_size; k++)
				{
					for (int l = 0; l < 32; l++)
					{
						if ((gpu_et[j][k] & (1 << l)))
						{
							gpu_lp_ets[j] = k * 32 + l;
							find = true;
							break;
						}
					}
					if (find)
					{
						find ^= find;
						break;
					}
					else
						gpu_lp_ets[j] = -1;
				}
			}
		}
	}
}
//CUDA��ȥ�㷨
void CUDA_GE(bitset<column_num_c>* ek, bitset<column_num_c>* et, int* lp_ets)
{
	for (int i = 0; i < column_num_c; i++)
	{
		eks_bits[i] = new int[bit_size];
		for (int j = 0; j < bit_size; j++)
			eks_bits[i][j] = 0;
	}
	for (int i = 0; i < et_num_c; i++)
	{
		ets_bits[i] = new int[bit_size];
		for (int j = 0; j < bit_size; j++)
			ets_bits[i][j] = 0;
	}
	string inek, inet;
	stringstream ss_inek, ss_inet;
	ifstream inElimKey(dir + "elimkey.txt");  // ��Ԫ��
	ifstream inElimTar(dir + "elimtar.txt");  // ����Ԫ��
	int inek_loc, p_ek = 0, inet_loc, p_et = 0;  // �������ݶ���
	int lp = -1;
	while (true)  // ��ȡ��Ԫ��
	{
		getline(inElimKey, inek);
		ss_inek = stringstream(inek);
		while (ss_inek >> inek)
		{
			inek_loc = stoi(inek);
			if (lp == -1)
			{
				lp = column_num_c - inek_loc - 1;
				lp_eks_c[lp] = lp;
			}
			// ���set����
			eks_bits[lp][(column_num_c - inek_loc - 1) / 32] |= (1 << ((column_num_c - inek_loc - 1) % 32));
		}
		lp = -1;
		p_ek++;
		if (inek.empty())
			break;
	}
	while (true)  // ��ȡ����Ԫ��
	{
		getline(inElimTar, inet);
		ss_inet = stringstream(inet);
		while (ss_inet >> inet)
		{
			inet_loc = stoi(inet);
			if (lp == -1)
			{
				lp = column_num_c - inet_loc - 1;
				lp_ets_c[p_et] = lp;
			}
			ets_bits[p_et][(column_num_c - inek_loc - 1) / 32] |= (1 << ((column_num_c - inek_loc - 1) % 32));
		}
		lp = -1;
		p_et++;
		if (inet.empty())
			break;
	}
	inElimKey.close();
	inElimTar.close();
	hipError_t ret;//���ڴ����飬�� CUDA �ӿڵ��óɹ��᷵�� cudaSucess
	int** gpu_ets;
	int** gpu_eks;
	int** gpu_ets_bits = new int* [et_num_c];  // ��ŵ���ָ���׵�ַ
	int** gpu_eks_bits = new int* [column_num_c];  // ��ŵ���ָ���׵�ַ
	int* gpu_lp_ets;
	int gpu_et_size = et_num_c * sizeof(int*);
	int gpu_ek_size = column_num_c * sizeof(int*);
	int bitset_size = sizeof(int) * bit_size;
	int gpu_lp_ets_size = et_num_c * sizeof(int);
	for (int i = 0; i < et_num; i++)
	{
		int* bits;  // ��ʱ����Դ��ַ
		if (hipMalloc(&bits, gpu_et_size) != hipSuccess)
			printf("hipMalloc gpudata failed!\n");
		if (hipMemcpy(bits, ets_bits[i], bitset_size, hipMemcpyHostToDevice) != hipSuccess)
			printf("hipMemcpyHostToDevice failed!\n");
		gpu_ets_bits[i] = bits;
	}
	if (hipMalloc(&gpu_ets, gpu_et_size) != hipSuccess)
		printf("hipMalloc gpudata failed!\n");
	if (hipMemcpy(gpu_ets, gpu_ets_bits, gpu_et_size, hipMemcpyHostToDevice) != hipSuccess)
		printf("hipMemcpyHostToDevice failed!\n");
	for (int i = 0; i < column_num_c; i++)
	{
		int* bits;  // ��ʱ����Դ��ַ
		if (hipMalloc(&bits, gpu_ek_size) != hipSuccess)
			printf("hipMalloc gpudata failed!\n");
		if (hipMemcpy(bits, eks_bits[i], bitset_size, hipMemcpyHostToDevice) != hipSuccess)
			printf("hipMemcpyHostToDevice failed!\n");
		gpu_eks_bits[i] = bits;
	}
	if (hipMalloc(&gpu_eks, gpu_ek_size) != hipSuccess)
		printf("hipMalloc gpudata failed!\n");
	if (hipMemcpy(gpu_eks, gpu_eks_bits, gpu_ek_size, hipMemcpyHostToDevice) != hipSuccess)
		printf("hipMemcpyHostToDevice failed!\n");
	//�����Դ�ռ䲢�ҽ��д�����
	if (hipMalloc(&gpu_lp_ets, gpu_lp_ets_size) != hipSuccess)
		printf("hipMalloc gpudata failed!\n");
	//�����ݴ����� GPU �˲����д�����
	if (hipMemcpy(gpu_lp_ets, lp_ets_c, gpu_lp_ets_size, hipMemcpyHostToDevice) != hipSuccess)
		printf("hipMemcpyHostToDevice failed!\n");
	dim3 dimBlock(BLOCK_SIZE, 1), dimGrid(1, 1); //�߳̿顢�߳�����
	hipEvent_t start, stop;  //��ʱ��
	float elapsedTime = 0.0;
	hipEventCreate(&start), hipEventCreate(&stop);
	hipEventRecord(start, 0);  //��ʼ��ʱ
	hipError_t exec;
	for (int i = 0; i < column_num_c; i++)  // ȡÿ����Ԫ�ӣ��Ա���Ԫ�н��в��������ڲ��л�
	{
		if (!(eks_bits[i][i / 32] & (1 << (i % 32))))  // ��Ԫ�ӱ������ʼ��ʱ���㡰�кš� = �����������
			upshift_kernel << <1, 1 >> > (i, gpu_eks, gpu_ets, gpu_lp_ets, et_num_c);
		hipDeviceSynchronize();//CPU �� GPU ֮���ͬ������
		exec = hipGetLastError();
		if (exec != hipSuccess)
			printf("upshift_kernel failed, %s\n", hipGetErrorString(exec));
		elim_kernel << <dimGrid, dimBlock >> > (i, gpu_eks, gpu_ets, gpu_lp_ets, et_num_c, column_num_c, bit_size);//������ȥ����ĺ˺���
		hipDeviceSynchronize();//CPU �� GPU ֮���ͬ������
		exec = hipGetLastError();
		if (exec != hipSuccess)
			printf("elim_kernel failed, %s\n", hipGetErrorString(exec));
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);//ֹͣ��ʱ
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("CUDA_GE:%f ms\n", elapsedTime);
	hipError_t cudaStatus2 = hipGetLastError();
	if (hipGetLastError() != hipSuccess)
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus2));
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
int main()
{
	cout << "�����СΪ" << column_num_c << "����Ԫ�Ӹ���Ϊ" << ek_num_c << "������Ԫ������Ϊ" << et_num_c << endl;
	CUDA_GE(eks_c, ets_c, lp_ets_c);
}